
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define MAX_SIZE 10000

#define GET_TIME(now){ \
	struct timeval t; \
	gettimeofday(&t, NULL); \
	now = t.tv_sec * 1000 + t.tv_usec / 1000.0; \
}

int arr[MAX_SIZE];
int brr[MAX_SIZE];

__global__ void mp(int *a, int *b)
{
	extern __shared__ int srr[];
	int i;
	int id_0 = blockIdx.x * blockDim.x;
	int id_1;

	srr[threadIdx.x] = a[id_0 + threadIdx.x];
	__syncthreads();

	for(i = 1 ;; i=i*2){
		if(i >= blockDim.x)break;
		id_1 = i * 2 * threadIdx.x;
		if(id_1 + i < blockDim.x){
			if(srr[id_1] < srr[id_1 + i]){
				srr[id_1] = srr[id_1 + i];
			}
		}
		__syncthreads();
	}
	if(threadIdx.x == 0)b[blockIdx.x] = srr[0];

}

int main(void) {

	int *d_a, *d_b;
	int block_size;
	int size = sizeof(int) * MAX_SIZE;
	int i, n, num;
	double start_time, end_time;

	FILE *fp = fopen("array.txt","r");

	printf("Input the Block Size: ");
	scanf("%d",&block_size);

	for(i = 0; i < MAX_SIZE; i++){
		fscanf(fp,"%d",&arr[i]);
	}

	fclose(fp);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);

	hipMemcpy(d_a, arr, size, hipMemcpyHostToDevice);

	GET_TIME(start_time);
	dim3 dimBlock (block_size, 1);
	if(MAX_SIZE / dimBlock.x != 0)n = 1;
	else{
		n = 0;
	}
	size = sizeof(int) * 1000;
	dim3 dimGrid(MAX_SIZE / dimBlock.x + n, 1);
	mp<<<dimGrid, dimBlock, size>>>(d_a,d_b);
	GET_TIME(end_time);

	hipMemcpy(brr, d_b, size, hipMemcpyDeviceToHost);

	num = 0;
	for(i = 0; i < MAX_SIZE / block_size + n; i++){
		if(num < brr[i]){
			num = brr[i];
		}
	}

	printf("THE MAXIMUM NUMBER IS %d\n",num);
	printf("VER D: Elapsed time is %e (msec)\n",end_time - start_time);

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}
