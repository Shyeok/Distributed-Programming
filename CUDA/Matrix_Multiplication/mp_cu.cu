
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define MAX_SIZE 1024

#define GET_TIME(now){ \
	struct timeval t; \
	gettimeofday(&t, NULL); \
	now = t.tv_sec * 1000 + t.tv_usec / 1000.0; \
}

int arr[MAX_SIZE * MAX_SIZE];
int brr[MAX_SIZE * MAX_SIZE];
int crr[MAX_SIZE * MAX_SIZE];

__global__ void mp(int *a, int *b, int *c)
{
	int i, sum;
	int id_0, id_1, id_2;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	sum = 0;

	for(i = 0; i < MAX_SIZE; i++){
		id_0 = row * MAX_SIZE + i;
		id_1 = i * MAX_SIZE + col;
		sum += a[id_0] * b[id_1];
	}

	id_2 = row * MAX_SIZE + col;
	c[id_2] = sum;
}

int main(void) {

	int *d_a, *d_b, *d_c;
	int block_size;
	int size = sizeof(int) * MAX_SIZE * MAX_SIZE;
	int i, j;
	double start_time, end_time;

	FILE *fp = fopen("matrix.txt","r");

	printf("Input the Block Size: ");
	scanf("%d",&block_size);

	for(i = 0; i < MAX_SIZE; i++){
		for(j = 0; j < MAX_SIZE; j++){
			fscanf(fp,"%d",&arr[i * MAX_SIZE + j]);
		}
	}

	for(i = 0; i < MAX_SIZE; i++){
		for(j = 0; j < MAX_SIZE; j++){
			fscanf(fp,"%d",&brr[i * MAX_SIZE + j]);
		}
	}

	fclose(fp);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, arr, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, brr, size, hipMemcpyHostToDevice);

	GET_TIME(start_time);
	dim3 dimBlock (block_size, block_size);
	dim3 dimGrid(MAX_SIZE / dimBlock.x, MAX_SIZE/dimBlock.y);
	mp<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
	GET_TIME(end_time);

	hipMemcpy(crr, d_c, size, hipMemcpyDeviceToHost);

	fp = fopen("matrix_cu.txt","w");
	for(i = 0; i < MAX_SIZE; i++){
		for(j = 0; j < MAX_SIZE; j++){
			fprintf(fp,"%d ",crr[i * MAX_SIZE + j]);
		}
		fprintf(fp,"\n");
	}
	fclose(fp);

	printf("Matrix Mulitplication Cuda VER: Elapsed time is %e (msec)\n",end_time - start_time);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
